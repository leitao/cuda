
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

__global__
void array_sum(float *x, float *y, float *z){
	int tid = blockIdx.x;
	if (tid < N)
		z[tid] = x[tid] * y[tid];
}
	

int main(void){
	float *x, *xx;
	float *y, *yy;
	float *z, *zz;

	// Allocate memory in the main memory
	xx = (float *) malloc(N*sizeof(float));
	yy = (float *) malloc(N*sizeof(float));
	zz = (float *) malloc(N*sizeof(float));

	// Add some data to this array
	for (int i=0 ; i < N; i++){
		xx[i] = 0.1*i;
		yy[i] = 1;
	}

	// Allocate memory in the GPU
	hipMalloc(&x, N*sizeof(float));
	hipMalloc(&y, N*sizeof(float));
	hipMalloc(&z, N*sizeof(float));

	// Copy memory from main memory to GPU
	hipMemcpy(x, xx, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(y, yy, N*sizeof(float), hipMemcpyHostToDevice);

	array_sum<<<1, N>>>(x, y, z);

	hipMemcpy(zz, z, N*sizeof(float), hipMemcpyDeviceToHost);
	
	for (int i=0 ; i < N; i++){
		printf("%d %20f\n", i, z[i]);
	}

	return 0;		
}
